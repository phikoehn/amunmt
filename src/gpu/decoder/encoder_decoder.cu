#include <iostream>

#include "common/god.h"

#include "encoder_decoder.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/dl4mt/dl4mt.h"
#include "gpu/decoder/encoder_decoder_state.h"
#include "gpu/decoder/best_hyps.h"

using namespace std;

namespace amunmt {
namespace GPU {

////////////////////////////////////////////
std::string EncoderDecoderState::Debug() const
{
	return states_.Debug();
}

mblas::Matrix& EncoderDecoderState::GetStates() {
  return states_;
}

mblas::Matrix& EncoderDecoderState::GetEmbeddings() {
  return embeddings_;
}

const mblas::Matrix& EncoderDecoderState::GetStates() const {
  return states_;
}

const mblas::Matrix& EncoderDecoderState::GetEmbeddings() const {
  return embeddings_;
}

////////////////////////////////////////////

EncoderDecoder::EncoderDecoder(
		const God &god,
		const std::string& name,
        const YAML::Node& config,
        size_t tab,
        const Weights& model)
  : Scorer(name, config, tab),
    model_(model),
    encoder_(new Encoder(model_)),
    decoder_(new Decoder(god, model_)),
    indices_(god.Get<size_t>("beam-size")),
    SourceContext_(new mblas::Matrix())
{}

void EncoderDecoder::Decode(const God &god, const State& in, State& out, const std::vector<size_t>& beamSizes) {
  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();

  decoder_->Decode(edOut.GetStates(),
                     edIn.GetStates(),
                     edIn.GetEmbeddings(),
                     *SourceContext_,
                     batchMapping_,
                     beamSizes);
}

State* EncoderDecoder::NewState() const {
  return new EDState();
}

void EncoderDecoder::BeginSentenceState(State& state, size_t batchSize) {
  EDState& edState = state.get<EDState>();
  decoder_->EmptyState(edState.GetStates(), *SourceContext_, batchSize, batchMapping_);
  decoder_->EmptyEmbedding(edState.GetEmbeddings(), batchSize);
}

void EncoderDecoder::SetSource(const Sentences& source) {
  encoder_->GetContext(source, tab_, *SourceContext_, batchMapping_);
}

void EncoderDecoder::AssembleBeamState(const State& in,
                               const Beam& beam,
                               State& out) {
  std::vector<size_t> beamWords;
  std::vector<size_t> beamStateIds;
  for (auto h : beam) {
     beamWords.push_back(h->GetWord());
     beamStateIds.push_back(h->GetPrevStateIndex());
  }

  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();
  indices_.resize(beamStateIds.size());
  thrust::host_vector<size_t> tmp = beamStateIds;
  mblas::copy_n(tmp.begin(), beamStateIds.size(), indices_.begin());

  mblas::Assemble(edOut.GetStates(), edIn.GetStates(), indices_);
  decoder_->Lookup(edOut.GetEmbeddings(), beamWords);
}

void EncoderDecoder::GetAttention(mblas::Matrix& Attention) {
  decoder_->GetAttention(Attention);
}

BaseMatrix& EncoderDecoder::GetProbs() {
  return decoder_->GetProbs();
}

mblas::Matrix& EncoderDecoder::GetAttention() {
  return decoder_->GetAttention();
}

size_t EncoderDecoder::GetVocabSize() const {
  return decoder_->GetVocabSize();
}

void EncoderDecoder::Filter(const std::vector<size_t>& filterIds) {
  decoder_->Filter(filterIds);
}

EncoderDecoder::~EncoderDecoder() {}

////////////////////////////////////////////
EncoderDecoderLoader::EncoderDecoderLoader(const std::string name,
                     const YAML::Node& config)
 : Loader(name, config) {}

void EncoderDecoderLoader::Load(const God &god) {
  std::string path = Get<std::string>("path");
  std::vector<size_t> devices = god.Get<std::vector<size_t>>("devices");

  size_t maxDeviceId = 0;
  for (size_t i = 0; i < devices.size(); ++i) {
    if (devices[i] > maxDeviceId) {
      maxDeviceId = devices[i];
    }
  }

  ThreadPool devicePool(devices.size());
  weights_.resize(maxDeviceId + 1);

  for(auto d : devices) {
    devicePool.enqueue([d, &path, this] {
      LOG(info) << "Loading model " << path << " onto gpu" << d;
      HANDLE_ERROR(hipSetDevice(d));
      weights_[d].reset(new Weights(path, d));
    });
  }
}

EncoderDecoderLoader::~EncoderDecoderLoader()
{
  for (size_t d = 0; d < weights_.size(); ++d) {
    const Weights *weights = weights_[d].get();
    if (weights) {
      HANDLE_ERROR(hipSetDevice(d));
      weights_[d].reset(nullptr);
    }
  }
}

ScorerPtr EncoderDecoderLoader::NewScorer(const God &god, const DeviceInfo &deviceInfo) const {
  //size_t i = deviceInfo.threadInd;
  size_t d = deviceInfo.deviceId; // TODO what is not using gpu0?
  //cerr << "NewScorer=" << i << " " << d << endl;

  HANDLE_ERROR(hipSetDevice(d));
  size_t tab = Has("tab") ? Get<size_t>("tab") : 0;
  return ScorerPtr(new EncoderDecoder(god, name_, config_,
                                      tab, *weights_[d]));
}

BestHypsBasePtr EncoderDecoderLoader::GetBestHyps(const God &god) const {
  return BestHypsBasePtr(new GPU::BestHyps(god));
}

}
}

