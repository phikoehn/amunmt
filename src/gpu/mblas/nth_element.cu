#include "hip/hip_runtime.h"
#include "gpu/mblas/nth_element.h"
#include <iostream>


namespace amunmt {
namespace GPU {

#define UNROLL_MAXARG_LOOP( n, max ) \
  if (tid < (n) && tid + (n) < ( max ) ) { \
    if (sdata[tid + ( n ) ] > sdata[tid]) { \
      sdata[tid] = sdata[tid + ( n ) ]; \
      indices[tid] = indices[tid + ( n ) ]; \
    } \
  }

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void gMaxElement(float* d_out, int* d_ind, float* d_in, int numBatches, int* batchFirstElementIdxs) {
  extern __shared__ float sdata[];
  __shared__ int indices[512];

  int tid = threadIdx.x;

  for (int batchIdx = 0; batchIdx < numBatches; ++batchIdx) {
    int begin = batchFirstElementIdxs[batchIdx];
    int end = batchFirstElementIdxs[batchIdx + 1];

    int i = begin + blockIdx.x * (blockDim.x * 2) + tid;

    sdata[tid] = -3.40282e+38f;

    if (i < end) {
      sdata[tid] = d_in[i];
      indices[tid] = i;
    }

    if (i + blockDim.x < end) {
      float a = d_in[i];
      float b = d_in[i + blockDim.x];
      if (a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while (i + 2 * gridDim.x * blockDim.x < end) {
      i += 2 * gridDim.x * blockDim.x;

      float a = d_in[i];
      if (a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if (i + blockDim.x < end) {
        float b = d_in[i + blockDim.x];
        if (b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for (int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if (tid < s && tid + s < end) {
        if (sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, end);
    UNROLL_MAXARG_LOOP(16, end);
    UNROLL_MAXARG_LOOP(8, end);
    UNROLL_MAXARG_LOOP(4, end);
    UNROLL_MAXARG_LOOP(2, end);
    UNROLL_MAXARG_LOOP(1, end);

    if (tid == 0) {
      d_out[blockIdx.x + batchIdx * gridDim.x] = sdata[0];
      d_ind[blockIdx.x + batchIdx * gridDim.x] = indices[0];
    }
    __syncthreads();
  }
}

__global__ void gMaxElementUpdate(float* binCosts, int* binIdxs, float* probs, int *batchFirstElements, float* outCosts, int* outIdxs, int *cummulatedBeamSizes, int NUM_BLOCKS) {
  extern __shared__ float sdata[];
  __shared__ int indices[512];
  __shared__ float bestBinCost;
  __shared__ int bestBinCostIdx;

  const int tid = threadIdx.x;
  const int batchIdx = blockIdx.x;
  const int N = batchFirstElements[batchIdx + 1] - batchFirstElements[batchIdx];
  int num_bins = int(N / (2 * 512)) + int(N % (2 * 512) != 0);
  if (num_bins > 500) {
    num_bins = 500;
  }

  for (int pos = cummulatedBeamSizes[batchIdx]; pos < cummulatedBeamSizes[batchIdx + 1]; ++pos) {
    int i = tid;

    sdata[tid] = -3.40282e+38f;

    if (i < num_bins) {
      sdata[tid] = binCosts[batchIdx * NUM_BLOCKS + i];
      indices[tid] = i;
    }

    if (i + blockDim.x < num_bins) {
      float a = binCosts[batchIdx * NUM_BLOCKS + i];
      float b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
      if (a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while (i + 2 * blockDim.x < num_bins) {
      i += 2 * blockDim.x;

      float a = binCosts[batchIdx * NUM_BLOCKS + i];
      if (a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if (i + blockDim.x < num_bins) {
        float b = binCosts[batchIdx * NUM_BLOCKS + i + blockDim.x];
        if (b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for (int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if (tid < s && tid + s < num_bins) {
        if (sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, num_bins);
    UNROLL_MAXARG_LOOP(16, num_bins);
    UNROLL_MAXARG_LOOP(8, num_bins);
    UNROLL_MAXARG_LOOP(4, num_bins);
    UNROLL_MAXARG_LOOP(2, num_bins);
    UNROLL_MAXARG_LOOP(1, num_bins);

    if (tid == 0) {
      bestBinCost = sdata[0];
      bestBinCostIdx = batchIdx * NUM_BLOCKS + indices[0];

      probs[binIdxs[bestBinCostIdx]] = -3.40282e+38f;

      outIdxs[pos] = binIdxs[bestBinCostIdx];
      outCosts[pos] = bestBinCost;
    }

    __syncthreads();

    i = batchFirstElements[batchIdx] + (bestBinCostIdx - batchIdx * NUM_BLOCKS) * (blockDim.x * 2) + tid;
    const int dist = num_bins * 2 * blockDim.x;

    sdata[tid] = -3.40282e+38f;

    if (i < batchFirstElements[batchIdx + 1]) {
      sdata[tid] = probs[i];
      indices[tid] = i;
    }

    if (i + blockDim.x < batchFirstElements[batchIdx + 1]) {
      float a = probs[i];
      float b = probs[i+blockDim.x];
      if (a > b) {
        sdata[tid] = a;
        indices[tid] = i;
      } else {
        sdata[tid] = b;
        indices[tid] = i + blockDim.x;
      }
    }

    while (i + dist < batchFirstElements[batchIdx + 1]) {
      i += dist;

      float a = probs[i];
      if (a > sdata[tid]) {
        sdata[tid] = a;
        indices[tid] = i;
      }

      if (i + blockDim.x < batchFirstElements[batchIdx + 1]) {
        float b = probs[i + blockDim.x];
        if (b > sdata[tid]) {
          sdata[tid] = b;
          indices[tid] = i + blockDim.x;
        }
      }
    }

    __syncthreads();

    for (int s = (blockDim.x >> 1); s > 32; s >>= 1) {
      if (tid < s && tid + s < batchFirstElements[batchIdx + 1]) {
        if (sdata[tid + s] > sdata[tid]) {
          sdata[tid] = sdata[tid + s];
          indices[tid] = indices[tid + s];
        }
      }
      __syncthreads();
    }

    UNROLL_MAXARG_LOOP(32, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(16, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(8, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(4, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(2, batchFirstElements[batchIdx + 1]);
    UNROLL_MAXARG_LOOP(1, batchFirstElements[batchIdx + 1]);

    if (tid == 0) {
      binCosts[bestBinCostIdx] = sdata[0];
      binIdxs[bestBinCostIdx] = indices[0];
    }
    __syncthreads();
  }
}

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n)
{
  int tid = threadIdx.x  + blockDim.x * blockIdx.x;
  if (tid < n) {
    int index = indeces[tid];
    d_out[tid] = d_in[index];
  }
}

NthElement::NthElement(size_t maxBeamSize, size_t maxBatchSize, hipStream_t& stream)
    : stream_(stream) ,
      NUM_BLOCKS(std::min(500, int(maxBeamSize * 85000 / (2 * BLOCK_SIZE)) + int(maxBeamSize * 85000 % (2 * BLOCK_SIZE) != 0)))
{
  //std::cerr << "NthElement::NthElement" << std::endl;

  HANDLE_ERROR( hipMalloc((void**)&d_ind, maxBatchSize * NUM_BLOCKS * sizeof(int)) );

  HANDLE_ERROR( hipMalloc((void**)&d_out, maxBatchSize * NUM_BLOCKS * sizeof(float)) );

  HANDLE_ERROR( hipMalloc((void**)&d_res_idx, maxBatchSize * maxBeamSize * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&d_res, maxBatchSize * maxBeamSize * sizeof(float)) );

  HANDLE_ERROR( hipHostAlloc((void**) &h_res, maxBeamSize * maxBatchSize* sizeof(float),
                              hipHostMallocDefault) );
  HANDLE_ERROR( hipHostAlloc((void**) &h_res_idx, maxBeamSize * maxBatchSize * sizeof(int),
                              hipHostMallocDefault) );

  HANDLE_ERROR( hipMalloc((void**)&d_breakdown, maxBeamSize * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void**)&d_batchPosition, (maxBatchSize + 1) * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&d_cumBeamSizes, (maxBatchSize + 1) * sizeof(int)) );
}

NthElement::~NthElement()
{
  HANDLE_ERROR(hipFree(d_ind));
  HANDLE_ERROR(hipFree(d_out));
  HANDLE_ERROR(hipFree(d_res_idx));
  HANDLE_ERROR(hipFree(d_res));
  HANDLE_ERROR(hipHostFree(h_res));
  HANDLE_ERROR(hipHostFree(h_res_idx));
  HANDLE_ERROR(hipFree(d_breakdown));
  HANDLE_ERROR(hipFree(d_batchPosition));
  HANDLE_ERROR(hipFree(d_cumBeamSizes));
}

void NthElement::getNBestList(float* probs, const std::vector<int>& batchFirstElementIdxs,
                              const std::vector<int>& cummulatedBeamSizes)
{
  HANDLE_ERROR( hipMemcpyAsync(d_batchPosition, batchFirstElementIdxs.data(), batchFirstElementIdxs.size() * sizeof(int),
                                hipMemcpyHostToDevice, stream_) );
  HANDLE_ERROR( hipMemcpyAsync(d_cumBeamSizes, cummulatedBeamSizes.data(), cummulatedBeamSizes.size() * sizeof(int),
                                hipMemcpyHostToDevice, stream_) );

  const int numBatches = batchFirstElementIdxs.size() - 1;

  gMaxElement<<<NUM_BLOCKS, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), stream_>>>
    (d_out, d_ind, probs, numBatches, d_batchPosition);

  gMaxElementUpdate<<<numBatches, BLOCK_SIZE, BLOCK_SIZE * sizeof(float), stream_>>>
    (d_out, d_ind, probs, d_batchPosition, d_res, d_res_idx, d_cumBeamSizes, NUM_BLOCKS);
}

void NthElement::getNBestList(const std::vector<size_t>& beamSizes, mblas::Matrix& Probs,
                  std::vector<float>& outCosts, std::vector<unsigned>& outKeys,
                  const bool isFirst) {
  std::vector<int> cummulatedBeamSizes(beamSizes.size() + 1, 0);
  std::vector<int> batchFirstElementIdxs(beamSizes.size() + 1, 0);

  const size_t vocabSize = Probs.Cols();
  for (size_t i = 0; i < beamSizes.size(); ++i) {

    cummulatedBeamSizes[i + 1] = cummulatedBeamSizes[i] + beamSizes[i];
    batchFirstElementIdxs[i + 1] += ((isFirst) ? (i + 1) : cummulatedBeamSizes[i + 1]) * vocabSize;
  }

  getNBestList(Probs.data(), batchFirstElementIdxs, cummulatedBeamSizes);
  GetPairs(cummulatedBeamSizes.back(), outKeys, outCosts);

}

void NthElement::GetPairs(size_t number,
                    std::vector<unsigned>& outKeys,
                    std::vector<float>& outValues) {

  HANDLE_ERROR( hipMemcpyAsync(h_res, d_res, number * sizeof(float),
                                hipMemcpyDeviceToHost, stream_) );
  HANDLE_ERROR( hipMemcpyAsync(h_res_idx, d_res_idx, number * sizeof(int),
                                hipMemcpyDeviceToHost, stream_) );
  hipStreamSynchronize(stream_);

  for (size_t i = 0; i < number; ++i) {
    outKeys.push_back(h_res_idx[i]);
    outValues.push_back(h_res[i]);
  }

  lastN = number;
}

void NthElement::getValueByKey(std::vector<float>& out, float* d_in) {
  gGetValueByKey<<<1, lastN, 0, stream_>>>
    (d_in, d_breakdown, h_res_idx, lastN);

  HANDLE_ERROR( hipMemcpyAsync(out.data(), d_breakdown, lastN * sizeof(float),
                                hipMemcpyDeviceToHost, stream_) );
  HANDLE_ERROR( hipStreamSynchronize(stream_));
}

}
}  // namespace GPU
