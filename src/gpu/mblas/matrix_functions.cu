#include "hip/hip_runtime.h"
#include "gpu/mblas/matrix_functions.h"

#include "gpu/mblas/handles.h"

namespace amunmt {
namespace GPU {
namespace mblas {

thread_local hipblasHandle_t* CublasHandler::handle_ = nullptr;
thread_local CudaStreamHandler* CudaStreamHandler::instance_ = nullptr;;

Matrix& Swap(Matrix& Out, Matrix& In) {
  size_t iRows = In.Rows();
  size_t iCols = In.Cols();
  size_t oRows = Out.Rows();
  size_t oCols = Out.Cols();

  Out.Reshape(iRows, iCols);
  In.Reshape(oRows, oCols);

  In.GetVec().swap(Out.GetVec());
  return Out;
}

__global__ void gMean(float* d_out, const float* d_in, const int* mapping,
                      int batchNum, int senLen, int stateLength) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < stateLength) {
    float sum = 0.0f;
    int counter = 0;

    for (int i = 0; i < batchNum * senLen; ++i) {
      sum += mapping[i] * d_in[i * stateLength + id];
      counter += mapping[i];

      if ((i + 1) % senLen == 0) {
        sum /= counter;
        d_out[(i / senLen) * stateLength + id] = sum;
        sum = 0.0f;
        counter = 0;
      }
    }
  }
}

void Mean(Matrix& Out, const Matrix& In, const DeviceVector<int>& mapping) {
  int batchNum = Out.Rows();
  int stateLength = Out.Cols();
  int sentenceLength = In.Rows() / batchNum;

  int nThreads = 512;
  int nBlocks =  (stateLength / 512) + ((stateLength % 512 == 0) ?  0 : 1);

  gMean<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (Out.data(), In.data(), thrust::raw_pointer_cast(mapping.data()),
     batchNum, sentenceLength, stateLength);
}

__global__ void gWeightedMean(float* d_out, const float* weights, const float* d_in, const int* mapping,
                              int numRows, int numCols, int srcLen) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < numRows * numCols) {
    int rowNo = id / numCols;
    int batchNo = mapping[rowNo];
    int statePos = id % numCols;

    float sum = 0.0f;
    for (int i = 0; i < srcLen; ++i) {
      sum += weights[rowNo * srcLen + i] * d_in[batchNo * srcLen * numCols + (i * numCols) + statePos];
    }

    d_out[id] = sum;
  }
}

void WeightedMean(Matrix& Out,const Matrix& Weights, const Matrix& In, const DeviceVector<int>& mapping) {
  int numRows = Weights.Rows();
  int numCols = In.Cols();

  Out.Resize(numRows, numCols);

  int nThreads = 512;
  int nBlocks =  (Out.size() / 512) + ((Out.size() % 512 == 0) ?  0 : 1);

  gWeightedMean<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (Out.data(), Weights.data(), In.data(), thrust::raw_pointer_cast(mapping.data()),
     numRows, numCols, Weights.Cols());
}

Matrix& Transpose(Matrix& Out, const Matrix& In) {
  size_t m = In.Rows();
  size_t n = In.Cols();

  Out.Resize(n, m);

  float alpha = 1.0;
  float beta  = 0.0;

  hipblasSgeam(CublasHandler::GetHandle(), HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, In.data(), n,
              &beta, In.data(), n, Out.data(), m);

  return Out;
}

Matrix& Transpose(Matrix& Out) {
  Matrix Temp;
  Transpose(Temp, Out);
  Swap(Out, Temp);
  return Out;
}

Matrix& Concat(Matrix& Out, const Matrix& In) {
  size_t oldSize = Out.size();
  Out.Resize(Out.Rows() + In.Rows(), Out.Cols());
  mblas::copy(In.begin(), In.end(), Out.begin() + oldSize);
  return Out;
}

Matrix& Copy(Matrix& Out, const Matrix& In) {
  Out.Resize(In.Rows(), In.Cols());
  mblas::copy(In.begin(), In.end(), Out.begin());
  return Out;
}

__global__ void gPasteRows(float* d_out, int outRows, int outCols, const float* d_in, int inRows, int inCols, int colNo, int sparse) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < inRows * inCols) {
    int inRow = id / inCols;
    int inCol = id % inCols;
    int outID = (outRows + sparse * inRow) * outCols + inCol + colNo;
    d_out[outID] = d_in[id];
  }
}
void PasteRows(Matrix& Out, const Matrix& In, const size_t rowNo, size_t colNo, size_t sparse) {
  int nColumns = In.Cols();
  int nRows = In.Rows();
  int nThreads = 512;
  int nBlocks =  (In.size() / 512) + ((In.size() % 512 == 0) ?  0 : 1);


  gPasteRows<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (Out.data(), rowNo, Out.Cols(), In.data(), In.Rows(), In.Cols(), colNo, sparse);
}

Matrix& PasteRow(Matrix& Out,
                 const Matrix& In,
                 const size_t r, const size_t c) {
  size_t start = r * Out.Cols() + c;
  mblas::copy(In.begin(), In.end(), Out.begin() + start);
  return Out;
}

Matrix& CopyRow(Matrix& Out,
                const Matrix& In,
                const size_t r, const size_t c) {
  size_t length = In.Cols() - c;
  Out.Resize(1, length);
  size_t start = r * In.Cols() + c;
  size_t end   = start + length;
  mblas::copy(In.begin() + start, In.begin() + end, Out.begin());
  return Out;
}

__global__ void gCopyRows(float* out, const float* in, size_t cols,
                          const size_t* targetRowIdx, size_t numPairs) {
  for (int bid = 0; bid < numPairs; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if (j < numPairs) {
      size_t dstId = j;
      size_t srcId = targetRowIdx[j];

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

Matrix& CopyRows(Matrix& Out,
                 const Matrix& In,
                 const size_t* dev,
                 size_t numPairs) {
  float* d_out = Out.data();
  const float* d_in = In.data();

  int threads = std::min(MAX_THREADS, (int)In.Cols());
  int blocks = std::min(MAX_BLOCKS, (int)numPairs);

  gCopyRows<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (d_out, d_in, In.Cols(), dev, numPairs);

  return Out;
}


Matrix& Assemble(Matrix& Out,
                 const Matrix& In,
                 const DeviceVector<size_t>& indeces) {
  Out.Resize(indeces.size(), In.Cols());
  CopyRows(Out, In, thrust::raw_pointer_cast(indeces.data()), indeces.size());
  return Out;
}

__global__ void gSlice(float* out, const float* in,
                       size_t n, size_t dim,
                       size_t rows, size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * dim;
      const float* rowIn = in + j * cols + n * dim;

      for(int tid = 0; tid < dim; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < dim)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

Matrix& Slice(Matrix& Out,
              const Matrix& In,
              size_t n, size_t dim) {

  Out.Resize(In.Rows(), dim);

  float* d_out = Out.data();
  const float* d_in = In.data();

  int threads = std::min(MAX_THREADS, (int)dim);
  int blocks = std::min(MAX_BLOCKS, (int)In.Rows());

  gSlice<<<blocks, threads, 0, CudaStreamHandler::GetStream()>>>
    (d_out, d_in, n, dim, In.Rows(), In.Cols());
  return Out;
}

Matrix& Prod(hipblasHandle_t handle, Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {
  Matrix::value_type alpha = 1.0;
  Matrix::value_type beta = 0.0;

  size_t m = A.Rows();
  size_t k = A.Cols();
  if(transA)
    std::swap(m, k);

  size_t l = B.Rows();
  size_t n = B.Cols();
  if(transB)
    std::swap(l, n);

  size_t lda = A.Cols();
  size_t ldb = B.Cols();
  size_t ldc = B.Cols();

  if(transB)
    ldc = B.Rows();

  C.Resize(m, n);

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  hipblasSgemm(handle, opB, opA,
              n, m, k, &alpha, B.data(), ldb, A.data(), lda, &beta, C.data(), ldc);
  return C;
}

Matrix& Prod(Matrix& C, const Matrix& A, const Matrix& B,
             bool transA, bool transB) {
  return Prod(CublasHandler::GetHandle(), C, A, B, transA, transB);
}

__global__ void gSoftMax(float* softMaxP, size_t rows, size_t cols,
                         const int* batchID,
                         int batchNum,
                         const int* srcMapping,
                         int srcNum) {
  extern __shared__ float _share[];

  int rowIdx =  blockIdx.x;

  while (rowIdx < rows) {
    float* row = softMaxP + rowIdx * cols;

    float* _max = _share;
    _max[threadIdx.x] = row[threadIdx.x];
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        float value = row[id];
        value *= srcMapping[ batchID[rowIdx] * srcNum + id ];
        if (value > _max[threadIdx.x]) {
          _max[threadIdx.x] = value;
        }
      }
    }

    int len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        if(_max[threadIdx.x + skip] > _max[threadIdx.x])
          _max[threadIdx.x] = _max[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }
    __syncthreads();
    float max = _max[0];
    __syncthreads();

    float* _sum = _share;// + blockDim.x;
    _sum[threadIdx.x] = 0.0f;
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        row[id] = __expf(row[id] - max);
        row[id] *= srcMapping[ batchID[rowIdx] * srcNum + id ];
        _sum[threadIdx.x] += row[id];
      }
    }

    __syncthreads();

    len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        _sum[threadIdx.x] += _sum[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }

    __syncthreads();

    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        row[id] /= _sum[0];
      }
    }
    __syncthreads();
    rowIdx += gridDim.x;
  }
}

Matrix& Softmax(Matrix& Out, const DeviceVector<int>& batchIds, const DeviceVector<int>& srcMapping,size_t srcSize) {
  int blocks = std::min(MAX_BLOCKS, (int)Out.Rows());
  int threads = std::min(MAX_THREADS, (int)Out.Cols());
  int shared = sizeof(float) * threads * 2;

  gSoftMax<<<blocks, threads, shared, CudaStreamHandler::GetStream()>>>
    (Out.data(), Out.Rows(), Out.Cols(),
     thrust::raw_pointer_cast(batchIds.data()), batchIds.size(),
     thrust::raw_pointer_cast(srcMapping.data()), srcSize);
  return Out;
}

__global__ void gLogSoftMax(float* softMaxP, size_t rows, size_t cols) {
  extern __shared__ float _share[];

  int rowIdx =  blockIdx.x;

  while (rowIdx < rows) {
    float* row = softMaxP + rowIdx * cols;

    float* _max = _share;
    _max[threadIdx.x] = row[threadIdx.x];
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        if (row[id] > _max[threadIdx.x]) {
          _max[threadIdx.x] = row[id];
        }
      }
    }

    int len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        if(_max[threadIdx.x + skip] > _max[threadIdx.x])
          _max[threadIdx.x] = _max[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }
    __syncthreads();
    float max = _max[0];
    __syncthreads();

    float* _sum = _share;// + blockDim.x;

    _sum[threadIdx.x] = 0.0f;
    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        row[id] = __expf(row[id] - max);
        _sum[threadIdx.x] += row[id];
      }
    }

    len = blockDim.x;
    while (len != 1) {
      __syncthreads();

      int skip = (len + 1) >> 1;
      if (threadIdx.x < (len >> 1)) {
        _sum[threadIdx.x] += _sum[threadIdx.x + skip];
      }
      len = (len + 1) >> 1;
    }

    __syncthreads();

    for (int tid = 0; tid < cols; tid += blockDim.x) {
      int id = tid + threadIdx.x;
      if (id < cols) {
        row[id] = __logf(row[id]/_sum[0]);
      }
    }
    __syncthreads();
    rowIdx += gridDim.x;
  }
}


Matrix& LogSoftmax(Matrix& Out) {
  int blocks = std::min(MAX_BLOCKS, (int)Out.Rows());
  int threads = std::min(MAX_THREADS, (int)Out.Cols());
  int shared = sizeof(float) * threads * 2;

  gLogSoftMax<<<blocks, 500, shared, CudaStreamHandler::GetStream()>>>
    (Out.data(), Out.Rows(), Out.Cols());

  return Out;
}

__global__ void gSetColumn(float* d_in, int n_columns, int n_rows, int noColumn, float value) {
  int rowNumber = threadIdx.x  + blockDim.x * blockIdx.x;
  int index = noColumn + rowNumber * n_columns;

  if (index < n_columns * n_rows) {
    d_in[index] = value;
  }
}

void SetColumn(Matrix& In, int noColumn, float value) {
  int nColumns = In.Cols();
  int nRows = In.Rows();
  int nBlocks = nRows / 512 + ((nRows % 512 == 0) ?  0 : 1);
  int nThreads = std::min(512, nRows);

  gSetColumn<<<nBlocks, nThreads, 0, mblas::CudaStreamHandler::GetStream()>>>
    (In.data(), nColumns, nRows, noColumn, value);
}

__global__ void gFill(float* d_in, int size, float val) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < size) {
    d_in[index] = val;
  }
}

void Fill(Matrix& In, float value) {
  size_t size = In.size();
  int nThreads = std::min(512, (int)size);
  int nBlocks = (size / nThreads) + ((size % nThreads == 0) ? 0 : 1);

  gFill<<<nBlocks, nThreads, 0, CudaStreamHandler::GetStream()>>>
    (In.data(), size, value);
}

__global__
void gMapMatrix(float* d_in, int numRows, int numCols, int mappingCols, const int* mapping, int i) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < numRows * numCols) {
    int batchIdx = tid / numCols;
    d_in[tid] *= mapping[mappingCols * batchIdx + i];
  }
}

void MapMatrix(Matrix& state, const DeviceVector<int>& mapping, size_t i) {
  int batchSize = state.Rows();
  int stateLength = state.Cols();
  int sentenceLength = mapping.size() / batchSize;

  int numThreads = std::min((int)state.size(), 512);
  int numBlocks = (state.size() / numThreads) + 1;

  float* d_in = thrust::raw_pointer_cast(state.data());
  const int* d_mapping = thrust::raw_pointer_cast(mapping.data());

  gMapMatrix<<<numBlocks, numThreads, 0, CudaStreamHandler::GetStream()>>>
    (d_in, batchSize, stateLength, sentenceLength, d_mapping, i);
}

}  // namespace mblas
}  // namespace GPU
}
