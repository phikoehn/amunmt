#include "hip/hip_runtime.h"
#include "encoder.h"

using namespace std;

namespace amunmt {
namespace GPU {

Encoder::Encoder(const Weights& model)
: embeddings_(model.encEmbeddings_),
  forwardRnn_(model.encForwardGRU_),
  backwardRnn_(model.encBackwardGRU_)
{}

size_t GetMaxLength(const Sentences& source, size_t tab) {
  size_t maxLength = source.at(0)->GetWords(tab).size();
  for (size_t i = 0; i < source.size(); ++i) {
    const Sentence &sentence = *source.at(i);
    maxLength = std::max(maxLength, sentence.GetWords(tab).size());
  }
  return maxLength;
}

std::vector<std::vector<size_t>> GetBatchInput(const Sentences& source, size_t tab, size_t maxLen) {
  std::vector<std::vector<size_t>> matrix(maxLen, std::vector<size_t>(source.size(), 0));

  for (size_t j = 0; j < source.size(); ++j) {
    for (size_t i = 0; i < source.at(j)->GetWords(tab).size(); ++i) {
        matrix[i][j] = source.at(j)->GetWords(tab)[i];
    }
  }

  return matrix;
}

void Encoder::GetContext(const Sentences& source, size_t tab, mblas::Matrix& Context,
                         DeviceVector<int>& dMapping) {
  size_t maxSentenceLength = GetMaxLength(source, tab);

  thrust::host_vector<int> hMapping(maxSentenceLength * source.size(), 0);
  for (size_t i = 0; i < source.size(); ++i) {
    for (size_t j = 0; j < source.at(i)->GetWords(tab).size(); ++j) {
      hMapping[i * maxSentenceLength + j] = 1;
    }
  }

  dMapping = hMapping;

  Context.Resize(maxSentenceLength * source.size(),
                 forwardRnn_.GetStateLength() + backwardRnn_.GetStateLength());

  auto input = GetBatchInput(source, tab, maxSentenceLength);

  for (size_t i = 0; i < input.size(); ++i) {
    if (i >= embeddedWords_.size()) {
      embeddedWords_.emplace_back();
    }
    embeddings_.Lookup(embeddedWords_[i], input[i]);
  }

  forwardRnn_.GetContext(embeddedWords_.cbegin(),
                         embeddedWords_.cbegin() + maxSentenceLength,
                         Context, source.size(), false);

  backwardRnn_.GetContext(embeddedWords_.crend() - maxSentenceLength,
                          embeddedWords_.crend() ,
                          Context, source.size(), true, &dMapping);
}

}
}

