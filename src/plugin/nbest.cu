#include "hip/hip_runtime.h"
#include "nbest.h"
#include <sstream>
#include <fstream>
#include <algorithm>

#include "common/utils.h"
#include "common/vocab.h"

NBest::NBest(
    const std::string& srcPath,
    const std::string& nbestPath,
    const std::shared_ptr<Vocab> srcVocab,
    const std::shared_ptr<Vocab> trgVocab,
    const size_t maxBatchSize)
    : srcVocab_(srcVocab),
      trgVocab_(trgVocab),
      maxBatchSize_(maxBatchSize) {
  ParseInputFile(srcPath);
  Parse_(nbestPath);
}

NBest::NBest(
  const std::shared_ptr<Vocab> srcVocab,
  const std::shared_ptr<Vocab> trgVocab,
  const std::vector<std::string>& nBestList,
  const size_t maxBatchSize)
    : srcVocab_(srcVocab),
      trgVocab_(trgVocab),
      maxBatchSize_(maxBatchSize) {
  for (size_t i = 0; i < nBestList.size(); ++i) {
    std::vector<std::string> tokens;
    Split(nBestList[i], tokens);
    data_.push_back(tokens);
  }
}

void NBest::ParseInputFile(const std::string& path) {
    std::ifstream file(path);
    srcSentences_.clear();
    std::string line;
    while (std::getline(file, line).good()) {
      Trim(line);
      srcSentences_.push_back(line);
    }
}

std::string NBest::GetSentence(const size_t index) const {
  return srcSentences_[index];
}

std::vector<std::string> NBest::GetTokens(const size_t index) const {
  std::vector<std::string> tokens;
  Split(srcSentences_[index], tokens);
  return tokens;
}

std::vector<size_t> NBest::GetEncodedTokens(const size_t index) const {
  std::vector<std::string> tokens;
  Split(srcSentences_[index], tokens);
  return srcVocab_->Encode(tokens, true);
}

void NBest::Parse_(const std::string& path) {
  std::ifstream file(path);

  std::string line;
  size_t lineCount = 0;
  indexes_.push_back(lineCount);

  while (std::getline(file, line).good()) {
    boost::trim(line);
    std::vector<std::string> fields;
    Split(line, fields, " ||| ");
    if (lineCount && (data_.back()[0] != fields[0])) {
      indexes_.push_back(lineCount);
    }
    data_.push_back(fields);
    ++lineCount;
  }
  indexes_.push_back(data_.size());
}


inline std::vector< std::vector< std::string > > NBest::SplitBatch(std::vector<std::string>& batch) const {
  std::vector< std::vector< std::string > > splittedBatch;
  for (auto& sentence : batch) {
    Trim(sentence);
    std::vector<std::string> words;
    Split(sentence, words);
    splittedBatch.push_back(words);
  }
  return splittedBatch;
}

inline NBestBatch NBest::EncodeBatch(const std::vector<std::vector<std::string> >& batch) const {
  NBestBatch encodedBatch;
  for (auto& sentence: batch) {
    encodedBatch.push_back(trgVocab_->Encode(sentence, true));
  }
  return encodedBatch;
}

inline NBestBatch NBest::MaskAndTransposeBatch(const NBestBatch& batch) const {
  size_t maxLength = 0;
  for (auto& sentence: batch) {
    maxLength = max(maxLength, sentence.size());
  }
  NBestBatch masked;
  for (size_t i = 0; i < maxLength; ++i) {
      masked.emplace_back(batch.size(), 0);
      for (size_t j = 0; j < batch.size(); ++j) {
          if (i < batch[j].size()) {
              masked[i][j] = batch[j][i];
          }
      }
  }
  return masked;
}


NBestBatch NBest::ProcessBatch(std::vector<std::string>& batch) const {
  return MaskAndTransposeBatch(EncodeBatch(SplitBatch(batch)));
}

NBestBatch NBest::ProcessBatch(std::vector<std::vector<std::string> >& batch) const {
  return MaskAndTransposeBatch(EncodeBatch(batch));
}

std::vector<NBestBatch> NBest::GetBatches(const size_t index) const {
  std::vector<NBestBatch> batches;
  std::vector<std::string> sBatch;
  for (size_t i = indexes_[index]; i <= indexes_[index+1]; ++i) {
    if (sBatch.size() == maxBatchSize_ || i == indexes_[index+1]) {
      batches.push_back(ProcessBatch(sBatch));
      sBatch.clear();
      if (i == indexes_[index+1]) {
        break;
      }
    }
    sBatch.push_back(data_[i][1]);
  }
  return batches;
}

std::vector<NBestBatch> NBest::DivideNBestListIntoBatches() const {
  std::vector<NBestBatch> batches;
  std::vector<std::vector<std::string> > sBatch;
  for (size_t i = 0; i < data_.size(); ++i) {
    sBatch.push_back(data_[i]);
    if (sBatch.size() == maxBatchSize_ || i == data_.size() - 1) {
      batches.push_back(ProcessBatch(sBatch));
      sBatch.clear();
    }
  }
  return batches;
}


size_t NBest::size() const {
  return indexes_.size() - 1;
}
